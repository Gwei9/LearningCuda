﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


int main()
{
    hipDeviceProp_t prop;

    int count;

    hipError_t cudaStatus = hipGetDeviceCount(&count);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "getDeviceCount failed!");
    }

    for (int i = 0; i < count; ++i) {
        cudaStatus = hipGetDeviceProperties(&prop, i);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "getDeviceProperties failed!");
        }

        printf("  --- General Information for device %d ---\n", i);
        printf("Name:  %s\n", prop.name);
        printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
        printf("Device copy overlap:  ");
        if (prop.deviceOverlap)
            printf("Enabled\n");
        else
            printf("Disabled\n");

        printf("  --- Memory Information for device %d ---\n", i);
        printf("Total global Mem: %1d\n", prop.totalGlobalMem);
        printf("Total constant Mem: %1d\n", prop.totalConstMem);
        printf("\n");
    }

    return 0;
}
