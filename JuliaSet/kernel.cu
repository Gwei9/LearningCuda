#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "cpu_bitmap.h"


#define DIM 1000

struct hipComplex {
    float r;
    float i;
    hipComplex(float a, float b) : r(a), i(b) {}
    __device__ float magnitude2(void) {
        return r * r + i * i;
    }
    __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
    }
    __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r + a.r, i + a.i);
    }
};

__device__ int julia(int x, int y) {
    const float scale = 1.5;
    float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
    float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    for (int i = 0; i < 200; ++i) {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}

__global__ void kernel(unsigned char* ptr) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;

    int juliaValue = julia(x, y);
    ptr[offset * 4 + 0] = 255 * juliaValue;
    ptr[offset * 4 + 1] = 0;
    ptr[offset * 4 + 2] = 0;
    ptr[offset * 4 + 3] = 255;
}

int main()
{
    CPUBitmap bitmap(DIM, DIM);
    unsigned char* dev_bitmap;

    hipError_t cudaStatus;
    cudaStatus = hipMalloc((void**)&dev_bitmap, bitmap.image_size());
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    dim3 grid(DIM, DIM);
    kernel << <grid, 1 >> > (dev_bitmap);

    cudaStatus = hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpyDeviceToHost failed!");
    }

    bitmap.display_and_exit();

    cudaStatus = hipFree(dev_bitmap);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFree failed!");
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
